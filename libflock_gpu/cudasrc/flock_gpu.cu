#include "hip/hip_runtime.h"
#include <iostream>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>


// Needed for output functions within the kernel
#include <stdio.h>

// printf() is only supported
// for devices of compute capability 2.0 and higher
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
   #define printf(f, ...) ((void)(f, __VA_ARGS__),0)
#endif

// For thrust routines (e.g. stl-like operators and algorithms on vectors)
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>


#include<sys/time.h>

// My own include function to generate some randomness
#include "random.cuh"

#include "flock_gpu.h"


/// The number of points to generate within 0,1
#define NUM_POINTS 10

/// The resolution of our grid (dependent on the radius of influence of each point)
#define GRID_RESOLUTION 4


/// The null hash indicates the point isn't in the grid (this shouldn't happen if your extents are correctly chosen)
#define NULL_HASH UINT_MAX

/// Used to define a point not in the neighbourhood
#define NULL_PNT UINT_MAX

/// Used to define a non neighbourhood cell
#define NULL_CELL UINT_MAX

/**
  * Find the cell hash of each point. The hash is returned as the mapping of a point index to a cell.
  * If the point isn't inside any cell, it is set to NULL_HASH. This may have repercussions later in
  * the code.
  * \param Px The array of x values
  * \param Py The array of y values
  * \param Pz the array of z values
  * \param hash The array of hash output
  * \param N The number of points (dimensions of Px,Py,Pz and hash)
  * \param res The resolution of our grid.
  */
__global__ void pointHash(unsigned int *hash,
                          const float *Px,
                          const float *Py,
                          const unsigned int N,
                          const unsigned int res) {
    // Compute the index of this thread: i.e. the point we are testing
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        // Note that finding the grid coordinates are much simpler if the grid is over the range [0,1] in
        // each dimension and the points are also in the same space.
        int gridPos[2];
        gridPos[0] = floor(Px[idx] * res);
        gridPos[1] = floor(Py[idx] * res);
        //gridPos[2] = floor(Pz[idx] * res);

        // Test to see if all of the points are inside the grid
        bool isInside = true;
        unsigned int i;
        for (i=0; i<2; ++i)
            if ((gridPos[i] < 0) || (gridPos[i] > res)) {
                isInside = false;
            }

        // Write out the hash value if the point is within range [0,1], else write NULL_HASH
        if (isInside) {
            hash[idx] = gridPos[0] + (gridPos[1] * res);
        } else {
            hash[idx] = NULL_HASH;
        }
        // Uncomment the lines below for debugging. Not recommended for 4mil points!
        //printf("pointHash<<<%d>>>: P=[%f,%f,%f] gridPos=[%d,%d,%d] hash=%d\n",
        //       idx, Px[idx], Py[idx], Pz[idx],
        //       gridPos[0], gridPos[1], gridPos[2], hash[idx]);
    }
}

/**
  * Compute the grid cell occupancy from the input vector of grid hash values. Note that the hash[]
  * vector doesn't need to be presorted, but performance will probably improve if the memory is
  * contiguous.
  * \param cellOcc A vector, size GRID_RES^3, which will contain the occupancy of each cell
  * \param hash A vector, size NUM_POINTS, which contains the hash of the grid cell of this point
  * \param nCells The size of the cellOcc vector (GRID_RES^3)
  * \param nPoints The number of points (size of hash)
  */
__global__ void countCellOccupancy(unsigned int *cellOcc,
                                   unsigned int *hash,
                                   unsigned int nCells,
                                   unsigned int nPoints) {
    // Compute the index of this thread: i.e. the point we are testing
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform a sanity check and then atomic add to the occupancy count of the relevant cell
    if ((idx < nPoints) && (hash[idx] < nCells)) {
        atomicAdd(&(cellOcc[hash[idx]]), 1);
    }
}


__device__ float distancePoints(float pntX,
                          float pntY,
                          unsigned int N,
                          float neighbourPntx,
                          float neighbourPnty)
{

    float distance = 0;

    distance = sqrt((pntX-neighbourPntx)*(pntX-neighbourPntx)+(pntY - neighbourPnty)*(pntY - neighbourPnty));


    return distance;
}


__global__ void neighbourhoodCells(unsigned int *neighbourCells,
                                   float neighbourhoodDist,
                                  const unsigned int res,
                                  unsigned int cell)
{

    // Compute the index of this thread: i.e. the point we are testing
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Compute the index of this thread: i.e. the point we are testing
    uint idy = blockIdx.y * blockDim.y + threadIdx.y;




    // the number of cells in each direction to check
    int bucketRadius = ceil(neighbourhoodDist/(1.0/float(res)));;

    // Find surrounding cells
    int y = floor(float(cell/res));
    int x = cell -(y*res);

    int count = 0;

    // add neighbour cells and current cell to array
    if(idx <= 2 * bucketRadius && idy <= 2 * bucketRadius)
    {
        //printf("idx: %d, idy %d \n", idx, idy);

        int i = x - bucketRadius + idx;
        int j = y - bucketRadius + idy;

        //printf("%d i, %d j \n", i , j);

        //printf("%d idx, %d idy \n", idx , idy);

        if(i>=0 && j>=0 && i<=res-1 && j<= res-1)
        {
            neighbourCells[(j*res) + i] = (j*res) + i;
            //printf(" %d neighbour cell added \n",neighbourCells[(j*res) + i]);

            //count ++;
        }

    }



//    // Remove empty cells
//    // go through cells
//    if(idx < (1 + (2*bucketRadius))*(1 + (2*bucketRadius)))
//    {
//        if(neighbourCells[idx]<res*res)
//        {
//            printf("%d neighbour cell", neighbourCells[idx]);
//            // if cell empty
//            if(cellOcc[neighbourCells[idx]] == 0)
//            {
//                //add points to neighbourhood

//                printf("deleting cell %d \n", neighbourCells[idx]);
//                neighbourCells[idx] = NULL_CELL;

//            }
//        }




//    }




//    for(int i = 0; i < count; i++)
//    {
//        // if cell empty
//        if(cellOcc[neighbourCells[i]] == 0)
//        {
//            //add points to neighbourhood

//            printf("deleting cell %d \n", neighbourCells[i]);
//            neighbourCells[i] = NULL_CELL;

//        }
//    }




//    for( int i = x - bucketRadius; i <= x + bucketRadius; ++i ){
//        for( int j = y - bucketRadius; j <= y + bucketRadius; ++j ){
//            if(i>=0 && j>=0 && i<=res-1 && j<= res-1)
//            {
//                    neighbourCells[count] = (j*res) + i;

//                    printf(" %d neighbour cell added \n",neighbourCells[count]);

//                    count ++;

//            }

//        }
//    }


}

__global__ void emptyCellCheck(unsigned int *neighbourCells,
                               unsigned int *cellOcc,
                              const unsigned int res)
{
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Remove empty cells
    if(idx < res*res)
    {
//        printf("%d index \n", idx);

        if(neighbourCells[idx]<res*res)
        {

            //printf("%d neighbour cell \n", neighbourCells[idx]);
            // if cell empty
            if(cellOcc[neighbourCells[idx]] == 0)
            {
                //add points to neighbourhood

                //printf("deleting cell %d \n", neighbourCells[idx]);
                neighbourCells[idx] = NULL_CELL;

            }
        }
    }

}





// EXTEND TO 3Dzz
// find cells surrounding current particles cells
__global__ void nearestNeighbourPnts(float *neighbourhood,
                                     unsigned int *neighbourCells,
                                     unsigned int *hash,
                                     const unsigned int N,
                                     const unsigned int res
                                )
{

    // Compute the index of this thread: i.e. the point we are testing
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint idy = blockIdx.y * blockDim.y + threadIdx.y;

    printf("%d x, %d y \n", idx, idy);

    if(idx<res*res)
    {
        // only check occupied cells
        if(neighbourCells[idx] < res*res)
        {
            if(idy < N)
            {
                if(neighbourCells[idx] == hash[idy] )
                {


                      neighbourhood[idy]=idy;

                }
            }


        }
    }

    // find points in cells
//    for(int i = 0; i < N; i++)
//    {
//        for(int j = 0; j<=count; j++)
//        {

//            if(hash[i] == neighbourCells[j])
//            {


//                  neighbourhoodX[count2]=i;
//                  neighbourhoodY[count2]=i;

//                  //neighbourhoodX[count2]=Px[i];
//                  //neighbourhoodY[count2]=Py[i];

//                  count2 ++;
//            }
//        }
//    }




//        int i = (x-bucketRadius) + idx;
//        int j = (y-bucketRadius) + idy;

//        if(i<=x + bucketRadius && j<=y + bucketRadius)
//        {
//            if((j*res + i) != cell)
//            {
//                neighbourhood[idy + idx] = (j*res) + i;

//            }
//        }


}



void FlockGPU::nearestNeighbour()
{

    // First thing is we'll generate a big old vector of random numbers for the purposes of
    // fleshing out our point data. This is much faster to do in one step than 3 seperate
    // steps.
    thrust::device_vector<float> d_Rand(NUM_POINTS*3);


    float * d_Rand_ptr = thrust::raw_pointer_cast(&d_Rand[0]);
    randFloats(d_Rand_ptr, NUM_POINTS*3);

    // We'll store the components of the 3d vectors in separate arrays.
    // This 'structure of arrays' (SoA) approach is usually more efficient than the
    // 'array of structures' (AoS) approach.  The primary reason is that structures,
    // like Float3, don't always obey the memory coalescing rules, so they are not
    // efficiently transferred to and from memory.  Another reason to prefer SoA to
    // AoS is that we don't aways want to process all members of the structure.  For
    // example, if we only need to look at first element of the structure then it
    // is wasteful to load the entire structure from memory.  With the SoA approach,
    // we can chose which elements of the structure we wish to read.
    thrust::device_vector<float> d_Px(d_Rand.begin(), d_Rand.begin()+NUM_POINTS);
    thrust::device_vector<float> d_Py(d_Rand.begin()+NUM_POINTS, d_Rand.begin()+2*NUM_POINTS);
//    thrust::device_vector<float> d_Pz(d_Rand.begin()+2*NUM_POINTS, d_Rand.end());

    // This vector will hold the grid cell occupancy (set to zero)
    thrust::device_vector<unsigned int> d_cellOcc(GRID_RESOLUTION*GRID_RESOLUTION, 0);

    // This vector will hold our hash values, one for each point
    thrust::device_vector<unsigned int> d_hash(NUM_POINTS);
    //thrust::copy(d_hash.begin(), d_hash.end(), std::ostream_iterator<unsigned int>(std::cout, " "));

    // Vector storing neighbour pnts idx
    thrust::device_vector<float> d_neighbours(NUM_POINTS,-1);

    // Holds neighbour cells
    thrust::device_vector<unsigned int> d_neighbourCells(GRID_RESOLUTION*GRID_RESOLUTION, NULL_CELL);

    // Typecast some raw pointers to the data so we can access them with CUDA functions
    unsigned int * d_hash_ptr = thrust::raw_pointer_cast(&d_hash[0]);
    unsigned int * d_cellOcc_ptr = thrust::raw_pointer_cast(&d_cellOcc[0]);
    float * d_neighbours_ptr = thrust::raw_pointer_cast(&d_neighbours[0]);
    unsigned int * d_neighbourCells_ptr = thrust::raw_pointer_cast(&d_neighbourCells[0]);
    float * d_Px_ptr = thrust::raw_pointer_cast(&d_Px[0]);
    float * d_Py_ptr = thrust::raw_pointer_cast(&d_Py[0]);
//   float * d_Pz_ptr = thrust::raw_pointer_cast(&d_Pz[0]);

    // The number of threads per blockshould normally be determined from your hardware, but 1024
    // is pretty standard. Remember that each block will be assigned to a single SM, with it's
    // own local memory.
    unsigned int nThreads = 1024;
    unsigned int nBlocks = NUM_POINTS / nThreads + 1;


    //dim3 threadsPerBlock(8, 8);
    //dim3 numBlocks(GRID_RESOLUTION/threadsPerBlock.x, GRID_RESOLUTION/threadsPerBlock.y);

     int blockDim = 1024 / GRID_RESOLUTION + 1; // 9 threads per block
     dim3 block(GRID_RESOLUTION, GRID_RESOLUTION); // block of (X,Y) threads
     dim3 grid(1, 1); // grid 2x2 blocks

     // for nearest neighbour
     unsigned int blockN = NUM_POINTS/ (GRID_RESOLUTION*GRID_RESOLUTION*NUM_POINTS) + 1;
     dim3 block2(GRID_RESOLUTION*GRID_RESOLUTION, NUM_POINTS); // block of (X,Y) threads
     dim3 grid2(blockN, blockN); // grid 2x2 blocks


    struct timeval tim;
    double t1, t2;
    gettimeofday(&tim, NULL);
    t1=tim.tv_sec+(tim.tv_usec/1000000.0);

    // The special CUDA syntax below executes our parallel function with the specified parameters
    // using the number of blocks and threads provided.
    pointHash<<<nBlocks, nThreads>>>(d_hash_ptr, d_Px_ptr, d_Py_ptr,
                                     NUM_POINTS,
                                     GRID_RESOLUTION);

    // Make sure all threads have wrapped up before completing the timings
    hipDeviceSynchronize();

    // Now we can sort our points to ensure that points in the same grid cells occupy contiguous memory
    thrust::sort_by_key(d_hash.begin(), d_hash.end(),
                        thrust::make_zip_iterator(
                            thrust::make_tuple( d_Px.begin(), d_Py.begin())));

    // Make sure all threads have wrapped up before completing the timings
    hipDeviceSynchronize();

    // Now we can count the number of points in each grid cell
    countCellOccupancy<<<nBlocks, nThreads>>>(d_cellOcc_ptr, d_hash_ptr, d_cellOcc.size(), d_hash.size());

    // Make sure all threads have wrapped up before completing the timings
    hipDeviceSynchronize();

    neighbourhoodCells<<<grid, block>>>(d_neighbourCells_ptr,0.24,GRID_RESOLUTION, 6);

    hipDeviceSynchronize();

    // sort into order
    thrust::sort(d_neighbourCells.begin(), d_neighbourCells.end());

    emptyCellCheck<<<nBlocks,nThreads>>>(d_neighbourCells_ptr, d_cellOcc_ptr,GRID_RESOLUTION);

    hipDeviceSynchronize();

    // sort into order again
    thrust::sort(d_neighbourCells.begin(), d_neighbourCells.end());

    // Finds pnt index in neighbourhood cells
    nearestNeighbourPnts<<<grid2, block2>>>(d_neighbours_ptr, d_neighbourCells_ptr, d_hash_ptr,NUM_POINTS,GRID_RESOLUTION);

    hipDeviceSynchronize();



    gettimeofday(&tim, NULL);
    t2=tim.tv_sec+(tim.tv_usec/1000000.0);
    std::cout << "Grid sorted "<<NUM_POINTS<<" points into grid of "<<GRID_RESOLUTION*GRID_RESOLUTION*GRID_RESOLUTION<<" cells in " << t2-t1 << "s\n";

    // Only dump the debugging information if we have a manageable number of points.
    if (NUM_POINTS <= 100) {
        thrust::copy(d_neighbourCells.begin(), d_neighbourCells.end(), std::ostream_iterator<unsigned int>(std::cout, " "));
        std::cout << "\n";
        thrust::copy(d_neighbours.begin(), d_neighbours.end(), std::ostream_iterator<float>(std::cout, " "));
        std::cout << "\n";
        thrust::copy(d_hash.begin(), d_hash.end(), std::ostream_iterator<unsigned int>(std::cout, " "));
        std::cout << "\n";
        thrust::copy(d_cellOcc.begin(), d_cellOcc.end(), std::ostream_iterator<unsigned int>(std::cout, " "));
    }
    //return 0;
}



