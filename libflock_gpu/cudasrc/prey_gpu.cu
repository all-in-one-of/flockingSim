#include "hip/hip_runtime.h"
#include "prey_gpu.cuh"
#include "flock_gpu.cuh"




Prey_GPU::Prey_GPU(Flock_GPU *_Flock, const int _ID) : Boid_GPU(_Flock, _ID)
{

}

Prey_GPU::~Prey_GPU()
{

}


void Prey_GPU::update()
{
    //std::cout<<m_pos.x<<" \n";




    flock();

    avoidBoundaries();

    //printf("vel: %f,%f,%f \n",m_vel[0], m_vel[1], m_vel[2]);






    m_pos+=m_vel;




    updateRotation();
}

void Prey_GPU::draw()
{


        glm::mat4 MV;
        glm::mat4 MVP;
        glm::mat3 N;



//        // translate to new position
//        MV = glm::translate(MV, m_pos);
//        MV = glm::rotate( MV, m_rotateAngle, glm::vec3( 0.0f, 1.0f, 0.0f ) );



//        MVP = m_Flock->getScene()->getProjection() * m_Flock->getScene()->getCamera().viewMatrix() * MV;

//        N = glm::mat3( glm::inverse( glm::transpose( MV ) ) );
//        // link matrices with shader locations
//        glUniformMatrix4fv( m_Flock->getScene()->getMVPAddress(), 1, GL_FALSE, glm::value_ptr( MVP ) );
//        glUniformMatrix4fv( m_Flock->getScene()->getMVAddress(), 1, GL_FALSE, glm::value_ptr( MV ) );

//        glUniformMatrix3fv( m_Flock->getScene()->getNAddress(), 1, GL_FALSE, glm::value_ptr( N ) );


//        // draw
//        glDrawArrays( GL_TRIANGLES, 0 , ( m_Flock->getScene()->getAmountVertexData() / 3 ) );


}

void Prey_GPU::avoidBoundaries()
{



    glm::vec3 desiredVel;

    if(m_pos.z >= 2 && m_vel.z >0)
    {
        desiredVel = {m_vel[0],0,-m_vel[2]};

        //printf("desired vel %f,%f,%f \n",desiredVel[0],desiredVel[1],desiredVel[2]);
        //std::cout<<" desired vel "<<desiredVel[0]<<" "<<desiredVel[2]<<" \n";

        //printf("vel: %f,%f,%f \n",m_vel[0],m_vel[1],m_vel[2]);
        m_vel += steerBoid(desiredVel);
        //printf("new vel: %f,%f,%f \n",m_vel[0],m_vel[1],m_vel[2]);

        //limitVel(0.02);
        //std::cout<<" out of z bounds\n";
    }
    else if(m_pos.z <= -2 && m_vel.z <0)
    {
        desiredVel = {m_vel[0],0,-m_vel[2]};

        //std::cout<<" desired vel "<<desiredVel[0]<<" "<<desiredVel[2]<<" \n";
        m_vel += steerBoid(desiredVel);

        //limitVel(0.02);
        //std::cout<<" out of -z bounds\n";
    }
    else if(m_pos.x >= 2 && m_vel.x >0)
    {
        desiredVel = {-m_vel[0],0,m_vel[2]};
        //std::cout<<" desired vel "<<desiredVel[0]<<" "<<desiredVel[2]<<" \n";
        m_vel += steerBoid(desiredVel);

        //imitVel(0.02);
        //std::cout<<" out of x bounds\n";
    }
    else if(m_pos.x <= -2 && m_vel.x <0)
    {
        desiredVel = {-m_vel[0],0,m_vel[2]};
        //std::cout<<" desired vel "<<desiredVel[0]<<" "<<desiredVel[2]<<" \n";
        m_vel += steerBoid(desiredVel);

        //limitVel(0.02);
        //std::cout<<" out of -x bounds\n";
    }


    //desiredVel /=

//    std::cout<<" desired vel "<<desiredVel[0]<<" "<<desiredVel[2]<<" \n";
//    m_vel += steerBoid(desiredVel);
    //m_vel = glm::normalize(m_vel);

    //limitVel(0.02);

}


void Prey_GPU::updateRotation()
{

    //rotation 0 when facing in z axis
        glm::vec3 facing = {0,0,1};

             //only update if moving
             if(m_vel != glm::vec3{0,0,0})
             {


                 float mag1 = glm::length(facing);
                 float mag2 = glm::length(m_vel);

                 //find angle between z axis and boids velocity vector
                 float steer = std::acos(glm::dot(facing, m_vel)/(mag1*mag2));

                 //convert from radians to degrees
                 //steer = steer*(180/M_PI);


                 //std::cout<<"vel "<<m_vel[0]<<"\n";
                 //std::cout<<"angle "<<steer<<" \n";


                 //if rotation past 180 degrees must take away from 360, then update boid rotation
                 if(m_vel[0]>0)
                 {
                     m_rotateAngle = steer;
                     m_rotation[1] = steer;
                 }
                 else
                 {
                     m_rotateAngle = 2*M_PI -steer;
                     m_rotation[1]= 360-steer;
                 }
             }

}

void Prey_GPU::flock()
{
        glm::vec3 steer = {0,0,0};

        //compute the flocking component vectors
        glm::vec3 alignment = {0,0,0};
        glm::vec3 cohesion = {0,0,0};
        glm::vec3 separation = {0,0,0};

        //alignment = alignBoid();
        //cohesion =cohesionBoid();
        separation =seperateBoid();


        //flocking component weights
        float alignmentWeight = 1;
        float cohesionWeight = 1;
        float separationWeight = 1.3;

        //find resulting flocking vector
        //steer[0] += (cohesion[0] * cohesionWeight) + (alignment[0] * alignmentWeight) + (separation[0] * separationWeight);
        //steer[2] += (cohesion[2] * cohesionWeight) + (alignment[2] * alignmentWeight) + (separation[2] * separationWeight);

        steer[0] += separation[0];
        steer[2] += separation[2];

        if(steer[0] != 0 && steer[2] != 0)
        {

            //steer =glm::normalize(steer);



            //steer towards flocking vector if required
            m_vel += steer;// steerBoid(steer);



        }


        if(m_vel != glm::vec3{0,0,0})
        {
            //m_vel = glm::normalize(m_vel);


            // limit velocity
            limitVel(0.02);
        }

}

glm::vec3 Prey_GPU::alignBoid()
{
    int numberOfNeighbours = 0;
    glm::vec3 alignmentVector {0,0,0};

    std::vector <Prey_GPU> boidsVector = m_Flock->getBoidsVector();

    float neighbourhoodRadius = 0.3;


    // find neighbour points of current boid in desired radius
//    nearestNeighbours(0.8f,m_Flock->getHashVec()[getID()]);

    m_Flock->findNeighbours(neighbourhoodRadius,getID());



    for(int i = 0; i<m_Flock->getNoBoids(); i++)
    {
        //std::cout<<m_Flock->getNeighbours()[i]<<"\n";


        // ignore pnt_max values
        if(m_Flock->getNeighbours()[i] <= m_Flock->getNoBoids())
        {
            if(boidsVector[m_Flock->getNeighbours()[i]].getID() != getID())
            {

                //std::cout<<getNeighbourPnts()[i]<< "neighbour points of "<<getID()<<" \n";
                if(distanceToBoid(boidsVector[m_Flock->getNeighbours()[i]]) < neighbourhoodRadius)
                {

                    alignmentVector[0] += boidsVector[m_Flock->getNeighbours()[i]].m_vel[0];
                    alignmentVector[2] += boidsVector[m_Flock->getNeighbours()[i]].m_vel[2];

                    numberOfNeighbours += 1;
                }
            }
        }


    }




//    for(int i = 0; i< m_Flock->getNoBoids(); i++)
//    {
//        //only flock with other flocking boids
//        if(boidsVector[i].getID() != getID())
//        {
//            if(boidsVector[i].m_flockFlag == true)
//            {
//                if( distanceToBoid(boidsVector[i]) < 0.3)
//                {






//                    alignmentVector[0] += boidsVector[i].m_vel[0];
//                    alignmentVector[2] += boidsVector[i].m_vel[2];

//                    //printf(" updated alignment vector: %f,%f,%f \n", alignmentVector[0],alignmentVector[1],alignmentVector[2]);
//                    numberOfNeighbours += 1;
//                }
//            }
//        }
//    }

    // avoid dividing by zero
    if(numberOfNeighbours != 0)
    {




        //find average velocity of boids in the current boids neighborhood
        alignmentVector[0] /= numberOfNeighbours;
        alignmentVector[2] /= numberOfNeighbours;





        //alignmentVector =  normaliseVector(alignmentVector); // glm::normalize(alignmentVector);


    }




    return alignmentVector;

}

glm::vec3 Prey_GPU::seperateBoid()
{
    int numberOfNeighbours = 0;
    glm::vec3 seperationVector {0,0,0};
    std::vector <Prey_GPU> boidsVector = m_Flock->getBoidsVector();

    glm::vec3 diff {0,0,0};

    float neighbourhoodRadius = 0.2;


    //std::cout<<getID()<<" point id \n";

    //std::cout<<m_Flock->getHashVec()[getID()]<<" cell id \n";

    // find neighbour points of current boid
    //nearestNeighbours(neighbourhoodRadius,getID());




//    m_Flock->findNeighbours(neighbourhoodRadius,getID());



//    for(int i = 0; i<m_Flock->getNoBoids(); i++)
//    {

//        //std::cout<<m_Flock->getHashVec()[getNeighbourPnts()[i]]<< " neighbour point cell \n";

//        // ignore pnt_max values
//        if(m_Flock->getNeighbours()[i] <= m_Flock->getNoBoids())
//        {
//            if(boidsVector[m_Flock->getNeighbours()[i]].getID() != getID())
//            {

//                //std::cout<<getNeighbourPnts()[i]<< "neighbour points of "<<getID()<<" \n";
//                if(distanceToBoid(boidsVector[m_Flock->getNeighbours()[i]]) < neighbourhoodRadius)
//                {

//                //std::cout<<"seperate \n";


//                //vector from current boid to neighbor
//                diff[0] = boidsVector[m_Flock->getNeighbours()[i]].m_pos[0]-m_pos[0];
//                diff[2] = boidsVector[m_Flock->getNeighbours()[i]].m_pos[2]-m_pos[2];

//                diff = glm::normalize(diff);

//                //the closer to its neighbors the greater the seperation vector
//                seperationVector[0] += diff[0] / (distanceToBoid(boidsVector[m_Flock->getNeighbours()[i]]));
//                seperationVector[2] += diff[2] / (distanceToBoid(boidsVector[m_Flock->getNeighbours()[i]]));


//                numberOfNeighbours += 1;
//                }
//            }
//        }


//    }






    for(int i = 0; i <m_Flock->getNoBoids(); i++)
    {
        if(boidsVector[i].getID() != getID())
        {
            if(boidsVector[i].m_flockFlag == true)
            {
                if(distanceToBoid(boidsVector[i]) <0.2)
                {

                    //vector from current boid to neighbor
                    diff[0] = boidsVector[i].m_pos[0]-m_pos[0];
                    diff[2] = boidsVector[i].m_pos[2]-m_pos[2];

                    glm::normalize(diff);

                    //the closer to its neighbors the greater the seperation vector
                    seperationVector[0] += diff[0] / (distanceToBoid(boidsVector[i]));
                    seperationVector[2] += diff[2] / (distanceToBoid(boidsVector[i]));


                    numberOfNeighbours += 1;
                }
            }
        }
    }

    //avoid dividing by zero
    if(numberOfNeighbours != 0)
    {

        seperationVector[0] /= numberOfNeighbours;
        seperationVector[2] /= numberOfNeighbours;

        //run in opposite direction to average neighbor position
        seperationVector[0] *= -1;
        seperationVector[2] *= -1;

        seperationVector = normaliseVector(seperationVector); //glm::normalize(seperationVector);


    }



    return seperationVector;

}

glm::vec3 Prey_GPU::cohesionBoid()
{
    int numberOfNeighbours = 0;
    glm::vec3 cohesionVector {0,0,0};

    std::vector <Prey_GPU> boidsVector = m_Flock->getBoidsVector();

//    std::cout<<getID()<<" point id \n";

//    std::cout<<m_Flock->getHashVec()[getID()]<<" cell id \n";

// spatial partitioning ---------------------------------------------------------------------

    // find neighbour points of current boid in desired radius
    //nearestNeighbours(1.0f,m_Flock->getHashVec()[getID()]);

    float neighbourhoodRadius = 0.4;

    m_Flock->findNeighbours(neighbourhoodRadius,getID());

    for(int i = 0; i<m_Flock->getNoBoids(); i++)
    {

        //std::cout<<m_Flock->getHashVec()[getNeighbourPnts()[i]]<< " neighbour point cell \n";

        // ignore pnt_max values
        if(m_Flock->getNeighbours()[i] <= m_Flock->getNoBoids())
        {
            if(boidsVector[m_Flock->getNeighbours()[i]].getID() != getID())
            {

                //std::cout<<getNeighbourPnts()[i]<< "neighbour points of "<<getID()<<" \n";
                if(distanceToBoid(boidsVector[m_Flock->getNeighbours()[i]]) < neighbourhoodRadius)
                {

                    cohesionVector[0] += boidsVector[m_Flock->getNeighbours()[i]].m_pos[0];
                    cohesionVector[2] += boidsVector[m_Flock->getNeighbours()[i]].m_pos[2];


                    numberOfNeighbours += 1;
                }

            }
        }


    }

    // slow code ----------------------------------------------------------------
//    for(int i = 0; i < m_Flock->getNoBoids(); i++)
//    {
//        if(boidsVector[i].getID() != getID())
//        {
//            if( boidsVector[i].m_flockFlag == true)
//            {
//                if(distanceToBoid(boidsVector[i]) < 0.4)
//                {




//                    cohesionVector[0] += boidsVector[i].m_pos[0];
//                    cohesionVector[2] += boidsVector[i].m_pos[2];


//                    numberOfNeighbours += 1;
//                }
//            }
//        }
//    }

    //avoid dividing by zero
    if(numberOfNeighbours != 0)
    {



        //find average position
        cohesionVector[0] /= numberOfNeighbours;
        cohesionVector[2] /= numberOfNeighbours;

        //find vector from agent to average position
        cohesionVector[0] = (cohesionVector[0] - m_pos[0]);
        cohesionVector[2] = (cohesionVector[2] - m_pos[2]);

        //std::cout<<cohesionVector[0]<<" "<<cohesionVector[2]<<" nomalise these\n";
        cohesionVector = normaliseVector(cohesionVector);// glm::normalize(cohesionVector);

    }

    return cohesionVector;


}

glm::vec3 Prey_GPU::steerBoid(glm::vec3 _target)
{



    glm::vec3 steerVec = {0.0f,0.0f,0.0f};

    glm::vec3 diff = {0.0f,0.0f,0.0f};


    diff[0] = _target[0] - m_vel[0];
    diff[2] = _target[2] - m_vel[2];

    //std::cout<<"steer "<<steer[0]<<steer[2]<<"\n";

    //printf("steer: %f,%f,%f \n",diff[0], diff[1], diff[2]);

    //printf("length: %f \n", vectorMagnitude(diff));

    steerVec[0] =( (diff[0]/vectorMagnitude(diff))*0.02f);
    steerVec[2] =( (diff[2]/vectorMagnitude(diff))*0.02f);



    //printf("new steer: %f,%f,%f \n",steer[0], steer[1], steer[2]);

    //std::cout<<steer[0]<<"\n";

    return steerVec;

}

float Prey_GPU::distanceToBoid(const Prey_GPU _boid)
{
    float distance = std::sqrt((m_pos[0]-_boid.m_pos[0])*(m_pos[0]-_boid.m_pos[0]) + (m_pos[2]-_boid.m_pos[2])*(m_pos[2]-_boid.m_pos[2]));

    return distance;

}

void Prey_GPU::limitVel(float _limit)
{


    if( glm::length(m_vel) > _limit)
    {

        m_vel[0] = (m_vel[0]/glm::length(m_vel))*_limit;
        m_vel[2] = (m_vel[2]/glm::length(m_vel))*_limit;

        //std::cout<<"new vel "<<m_vel[0]<<" \n";

    }
}


//void Prey_GPU::nearestNeighbours(float _neighbourhoodDist, int cell)
//{

//    //std::cout<<"nearest neighbour called on cell "<<cell<<" \n";

//    // divide by grid resolution as grid 0-1 and boids plane -3 - 3
//    _neighbourhoodDist /= (2 * m_Flock->m_gridRes);

//    // the number of cells in each direction to check
//    int bucketRadius = ceil(_neighbourhoodDist/(1.0/float(m_Flock->m_gridRes)));

//    // Find surrounding cells
//    int z = floor(float(cell/m_Flock->m_gridRes));
//    int x = cell -(z*m_Flock->m_gridRes);

//    int count = 0;

//    int neighbourCells[m_Flock->m_gridRes*m_Flock->m_gridRes];


//    for( int i = x - bucketRadius; i <= x + bucketRadius; ++i ){
//        for( int j = z - bucketRadius; j <= z + bucketRadius; ++j ){
//            if(i>=0 && j>=0 && i<=m_Flock->m_gridRes-1 && j<= m_Flock->m_gridRes-1)
//            {
//                //if((j*m_Flock->m_gridRes + i) != cell  )
//                //{
//                    neighbourCells[count] = (j*m_Flock->m_gridRes) + i;

//                    //std::cout<<neighbourCells[count]<<" neighbour cells \n";

//                    count ++;

//                //}
//            }

//        }
//    }



////    int count2;

//    // Remove empty cells
//    for(int i = 0; i < count; i++)
//    {
//        //std::cout<< neighbourCells[i]<<"\n";
//        // if cell  empty
//        if(m_Flock->getCellOcc()[neighbourCells[i]] == 0)
//        {
//            //add points to neighbourhood

//            //std::cout<< neighbourCells[i]<< " deleting cell \n";
//            neighbourCells[i] = -1;

////            count2++;

//        }

//    }


//    // clear neighbour points before recalculating
//    m_neighbourhoodPnts.clear();

//    int count2 = 0;
//    // order neighbours cells and iterate with while loop

//    // find points in cells
//    for(int i = 0; i < m_Flock->getNoBoids(); i++)
//    {
//        for(int j = 0; j<count; j++)
//        {
//            if(m_Flock->getHashVec()[i] == neighbourCells[j])
//            {
//                // add point id to list of points
//                m_neighbourhoodPnts.push_back(i);

//                count2++;

//            }
//        }

//    }

//}

glm::vec3 Prey_GPU::normaliseVector(glm::vec3 _vector)
{
    glm::vec3 normalisedVector {0,0,0};


    normalisedVector[0] = _vector[0] / _vector.length();
    normalisedVector[2] = _vector[2] / _vector.length();

    return normalisedVector;



}

float Prey_GPU::vectorMagnitude(glm::vec3 _vector)
{
    float mag;

    mag = std::sqrt((_vector[0]*_vector[0]) + (_vector[2]*_vector[2]));

    return mag;
}




